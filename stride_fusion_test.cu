#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "validate.h"
#include "common.h"
#include "blocksparse.h"
#include "fusion_blocksparse.h"


int main(){

    const uint batch_dim = 32;
    const uint heads_ = 4;
    const uint head_state_ = 128;
    const uint blk_size_= 32;
    const uint seq_len = 1024;
    const uint state = 512;
    const uint m = batch_dim*seq_len;
    const uint n = state;

    const uint nt_op = 0;
    const uint nn_op = 1; 
    const uint tn_op = 2;
    int count = 0;
    const uint ctx_blks_a_ = seq_len/blk_size_;
    const uint ctx_blks_b_ = seq_len/blk_size_;
    const uint ctx_blks_c_ = seq_len/blk_size_;
    const float scale = 1.0/sqrt(head_state_);


    const uint stride_lut_heads = 1;
    const uint stride_mask_heads = 1;
    uint stride_nn_max = 1;
    uint stride_tn_max = 1;
    const uint stride_attn_ctx = 32;

    uint stride_nt_lut_dim = 0;
    uint stride_nn_lut_dim = 0;
    uint stride_tn_lut_dim = 0;

    uint *layout = (uint*)malloc(sizeof(uint)*ctx_blks_a_*ctx_blks_a_);

    for(int i=0;i<(ctx_blks_a_*ctx_blks_a_);i++){

        layout[i] = 0;
    }

    uint n_bctx = seq_len/stride_attn_ctx;
    uint block_chucks = n_bctx/blk_size_;

    for(int i=0;i<ctx_blks_a_;i++){

        for(int j=0;j<ctx_blks_a_;j++){

            uint offset = i % block_chucks;

            if((j+offset >= i) && (j<=i)){

                layout[i*ctx_blks_a_+j] = 1;
                stride_nt_lut_dim++;
            }
        }
    }


    uint stride_blocks = stride_nt_lut_dim;
    stride_nn_lut_dim = stride_nt_lut_dim+ctx_blks_a_;
    stride_tn_lut_dim = stride_nn_lut_dim;

    uint2 *stride_nt_lut = (uint2*)malloc(sizeof(uint2)*(stride_nt_lut_dim));

    for(int i=0;i<stride_nt_lut_dim;i++){

        stride_nt_lut[i].x = 0;
        stride_nt_lut[i].y = 0;

    }


    count=0;
    for(int i=0;i<ctx_blks_a_;i++){
        for(int j=0;j<ctx_blks_a_;j++){

            if(layout[i*ctx_blks_a_+j] == 1){

                stride_nt_lut[count].x  = i;
                stride_nt_lut[count].y  = j;
                count++;
            }
        }
    }


    uint2 *d_stride_nt_lut;
    hipMalloc((void**)&d_stride_nt_lut, sizeof(uint2)*stride_nt_lut_dim);
    hipMemcpy(d_stride_nt_lut, stride_nt_lut, sizeof(uint2)*stride_nt_lut_dim, hipMemcpyHostToDevice);

    uint2 *stride_nn_lut = (uint2*)malloc(sizeof(uint2)*stride_nn_lut_dim);

    for(int i=0;i<stride_nn_lut_dim;i++){

        stride_nn_lut[i].x = 0;
        stride_nn_lut[i].y = 0;
    }


    for(int i=0;i<stride_nt_lut_dim;i++){

        stride_nn_lut[ctx_blks_a_+i].x = i;
        stride_nn_lut[ctx_blks_a_+i].y = stride_nt_lut[i].y;
        stride_nn_lut[stride_nt_lut[i].x].y++; 

    }

    stride_nn_lut[0].x = ctx_blks_a_;

    for(int i=1;i<ctx_blks_a_;i++){

        stride_nn_lut[i].x = stride_nn_lut[i-1].x + stride_nn_lut[i-1].y;

    }

    for(int i=0;i<ctx_blks_a_;i++){

        stride_nn_max = stride_nn_max > stride_nn_lut[i].y ? stride_nn_max : stride_nn_lut[i].y;

    }


    uint2 *d_stride_nn_lut;
    hipMalloc((void**)&d_stride_nn_lut, sizeof(uint2)*(stride_nn_lut_dim));
    hipMemcpy(d_stride_nn_lut, stride_nn_lut, sizeof(uint2)*stride_nn_lut_dim, hipMemcpyHostToDevice);


    bool *mask = (bool*)malloc(sizeof(bool)*stride_blocks*blk_size_*blk_size_);
    
    for(int i=0;i<(stride_blocks*blk_size_*blk_size_);i++){

        mask[i] = 1;
    }

    for(int i=0;i<stride_blocks;i++){

        for(int j=0;j<(blk_size_*blk_size_);j++){
            
            uint ii = j/blk_size_;
            uint jj = j%blk_size_;

            if((ii)<(jj)){
                mask[i*(blk_size_*blk_size_)+j] = 0;
            }
        }
    }

    unsigned int *stride_mask_np = (unsigned int*)malloc(sizeof(unsigned int)*stride_blocks*blk_size_);
    unsigned int *stride_mask = (unsigned int*)malloc(sizeof(unsigned int)*stride_blocks*blk_size_);

    for(int i=0;i<(stride_blocks*blk_size_);i++){
        stride_mask_np[i] = 0;
        for(int j=0;j<32;j++){
            bool keep = mask[i*blk_size_+j];
            unsigned int temp;
            temp = keep << (j);
            stride_mask_np[i] = stride_mask_np[i] | temp;  
        }
    }

    for(int i=0; i<stride_blocks; i++){
        for(int j=0; j<blk_size_; j++){
            stride_mask[j*stride_blocks+i] = stride_mask_np[i*blk_size_+j];
        }
    }

    unsigned int *sm_mask;
    hipMalloc((void**)&sm_mask, sizeof(unsigned int)*(stride_blocks*blk_size_));
    hipMemcpy(sm_mask, stride_mask_np, sizeof(unsigned int)*(stride_blocks*blk_size_), hipMemcpyHostToDevice);

    unsigned int *sm_mask1;
    hipMalloc((void**)&sm_mask1, sizeof(unsigned int)*(stride_blocks*blk_size_));
    hipMemcpy(sm_mask1, stride_mask, sizeof(unsigned int)*(stride_blocks*blk_size_), hipMemcpyHostToDevice);


    float *a_ptr_cpu = (float*)malloc(sizeof(float)*m*n);
    float *b_ptr_cpu = (float*)malloc(sizeof(float)*m*n);

    // char q_file[] = "/home/songshuhui/Desktop/Transformer-0523/comparison_precision/q";
    // char k_file[] = "/home/songshuhui/Desktop/Transformer-0523/comparison_precision/k";
    // readbinary(q_file, a_ptr_cpu, m*n);  
    // readbinary(k_file, b_ptr_cpu, m*n);

    for(int i=0;i<m*n;i++){

        a_ptr_cpu[i] = 1;
        b_ptr_cpu[i] = 1;

    }


    float *a_ptr, *b_ptr;
    hipMalloc((void**)&a_ptr, sizeof(float)*m*n);
    hipMalloc((void**)&b_ptr, sizeof(float)*m*n);

    hipMemcpy(a_ptr, a_ptr_cpu, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(b_ptr, b_ptr_cpu, sizeof(float)*m*n, hipMemcpyHostToDevice);

    

    uint2 *rblk_lutc = (uint2*)malloc(sizeof(uint2)*(stride_blocks));
    count = 0;
    for(int i=0;i<ctx_blks_a_;i++){
        for(int j=0;j<stride_nn_lut[i].y;j++){
            rblk_lutc[count].x = count;
            rblk_lutc[count].y = i;
            count++;
        }
    }

    uint2 *rblk_lut;
    hipMalloc((void**)&rblk_lut, sizeof(uint2)*stride_blocks);
    hipMemcpy(rblk_lut, rblk_lutc, sizeof(uint2)*(stride_blocks), hipMemcpyHostToDevice);

    float *y_ptr;
    hipMalloc((void**)&y_ptr, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_*blk_size_));
    
    // int *rblk_flag_c = (int*)malloc(sizeof(int)*ctx_blks_a_*batch_dim*heads_);
    // for(int i=0;i<(batch_dim*heads_);i++){
    //     for(int j=0;j<ctx_blks_a_;j++){
    //         rblk_flag_c[i*ctx_blks_a_+j] = stride_nn_lut[ctx_blks_a_-j-1].y;
    //     }
    // }

    int *rblk_flag;
    hipMalloc((void**)&rblk_flag, sizeof(int)*ctx_blks_a_*batch_dim*heads_);
    // hipMemcpy(rblk_flag, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);

    int *rblk_flag1;
    hipMalloc((void**)&rblk_flag1, sizeof(int)*ctx_blks_a_*batch_dim*heads_);
    // hipMemcpy(rblk_flag1, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);

    float *Max;
    hipMalloc((void**)&Max, sizeof(float)*batch_dim*heads_*stride_blocks*blk_size_);

    float *Sum;
    hipMalloc((void**)&Sum, sizeof(float)*batch_dim*heads_*stride_blocks*blk_size_);

    float *Maxc = (float*)malloc(sizeof(float)*batch_dim*heads_*stride_blocks*blk_size_);
    float *Sumc = (float*)malloc(sizeof(float)*batch_dim*heads_*stride_blocks*blk_size_);


    bhalf *c_ptr;
    hipMalloc((void**)&c_ptr, sizeof(bhalf)*(batch_dim*heads_*stride_blocks*blk_size_*blk_size_));
    bhalf *c_ptr_cpu = (bhalf*)malloc(sizeof(bhalf)*(batch_dim*heads_*stride_blocks*blk_size_*blk_size_));

    float *y1_ptr;
    hipMalloc((void**)&y1_ptr, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_*blk_size_));

    struct timeval GET_TIME_START, GET_TIME_END, GET_TIME_START1, GET_TIME_END1;

    double time=0, ntime=0, time1=0, time2=0;
    double time_avg = 0, ntime_avg = 0; 
    double time_min = 1000, ntime_min = 1000;


    double time_avg1 = 0, time_avg2 = 0;
    double time_min1 = 1000, time_min2 = 1000;

    int loops = 1000;

    hipStream_t custream;
    hipStreamCreate(&custream); 

    for(int i=0;i<loops;i++){

        fusion_attention1<float, float2>(custream, d_stride_nt_lut, a_ptr, b_ptr, c_ptr, d_stride_nn_lut, rblk_flag, rblk_flag1, rblk_lut,sm_mask, Max, Sum, y_ptr,
                        blk_size_, stride_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                        stride_lut_heads, stride_nt_lut_dim, stride_mask_heads, scale);

        // hipMemcpy(rblk_flag, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);
        // hipMemcpy(rblk_flag1, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);
        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_));

    }


    for(int i=0;i<loops;i++){

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START), NULL);

        
        fusion_attention1<float, float2>(custream, d_stride_nt_lut, a_ptr, b_ptr, c_ptr, d_stride_nn_lut, rblk_flag, rblk_flag1, rblk_lut, sm_mask, Max, Sum, y_ptr,
                            blk_size_, stride_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                            stride_lut_heads, stride_nt_lut_dim, stride_mask_heads, scale);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        time = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;
        time_avg += time;
        time_min = time_min > time ? time : time_min; 

        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_));

    }

    printf("fusion1_time_avg:%f \n", time_avg/loops);
    printf("fusion1_time_min:%f \n", time_min);   

    // float *yc = (float*)malloc(sizeof(float)*blk_size_*blk_size_*batch_dim*heads_*stride_blocks); 
    // hipMemcpy(yc, y_ptr, sizeof(float)*(stride_blocks*blk_size_*blk_size_), hipMemcpyDeviceToHost);

    // for(int i=0;i<32;i++){
    //     for(int j=0;j<blk_size_;j++){
    //         printf("%.15f ", yc[i*blk_size_*blk_size_+j*blk_size_]);
    //     }

    //     printf("\n");

    // }
    for(int i=0;i<loops;i++){

        fusion_attention2<float, float2>(custream, d_stride_nt_lut, a_ptr, b_ptr, c_ptr, d_stride_nn_lut, rblk_flag, rblk_flag1, sm_mask, Max, Sum, y_ptr,
                        blk_size_, stride_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                        stride_lut_heads, stride_nt_lut_dim, stride_mask_heads, scale);

        // hipMemcpy(rblk_flag, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);
        // hipMemcpy(rblk_flag1, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);
        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_));

    }

    time_avg = 0;
    time_min = 1000;

    for(int i=0;i<loops;i++){

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START), NULL);

        
        fusion_attention2<float, float2>(custream, d_stride_nt_lut, a_ptr, b_ptr, c_ptr, d_stride_nn_lut, rblk_flag, rblk_flag1, sm_mask, Max, Sum, y_ptr,
                            blk_size_, stride_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                            stride_lut_heads, stride_nt_lut_dim, stride_mask_heads, scale);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        time = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;
        time_avg += time;
        time_min = time_min > time ? time : time_min; 

        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*stride_blocks*blk_size_));

    }

    printf("fusion2_time_avg:%f \n", time_avg/loops);
    printf("fusion2_time_min:%f \n", time_min); 

    


    for(int i=0;i<loops;i++){

        bst_sgemm_nt(custream, d_stride_nt_lut, a_ptr, b_ptr, c_ptr, blk_size_, stride_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, stride_lut_heads, stride_nt_lut_dim);

        BlocksparseMaskedSoftmax<float,float2>(custream, d_stride_nn_lut, sm_mask1, c_ptr, y1_ptr, blk_size_, stride_blocks, batch_dim, heads_, ctx_blks_a_, stride_lut_heads, stride_nn_lut_dim, stride_nn_max, stride_mask_heads, scale);

    }
    
    for(int i=0;i<loops;i++){

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START1), NULL);
        gettimeofday(&(GET_TIME_START), NULL);
    
        bst_sgemm_nt(custream, d_stride_nt_lut, a_ptr, b_ptr, c_ptr, blk_size_, stride_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, stride_lut_heads, stride_nt_lut_dim);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        time1 = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START), NULL);      

        BlocksparseMaskedSoftmax<float,float2>(custream, d_stride_nn_lut, sm_mask1, c_ptr, y1_ptr, blk_size_, stride_blocks, batch_dim, heads_, ctx_blks_a_, stride_lut_heads, stride_nn_lut_dim, stride_nn_max, stride_mask_heads, scale);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        gettimeofday(&(GET_TIME_END1), NULL);


        time2 = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;
        ntime = (GET_TIME_END1.tv_sec - GET_TIME_START1.tv_sec) * 1000.0 + (GET_TIME_END1.tv_usec - GET_TIME_START1.tv_usec) / 1000.0;

        ntime_avg+=ntime;
        ntime_min = ntime_min > ntime ? ntime : ntime_min;

        time_avg1+=time1;
        time_min1 = time_min1 > time1 ? time1 : time_min1;

        time_avg2+=time2;
        time_min2 = time_min2 > time2 ? time2 : time_min2;

    }

    printf("sddmm_time_avg:%f \n", time_avg1/1000.0);
    printf("sddmm_time_min:%f \n", time_min1);

    printf("sfmx_time_avg:%f \n", time_avg2/1000.0);
    printf("sfmx_time_min:%f \n", time_min2);

    printf("nofusion_time_avg:%f \n", ntime_avg/1000.0);
    printf("nofusion_time_min:%f \n", ntime_min);

    // ntime_avg = 0;
    // ntime_min = 1000;

    // time_avg1 = 0;
    // time_min1 = 1000;

    // time_avg2 = 0;
    // time_min2 = 1000;

    // for(int i=0;i<1000;i++){

    //     bst_sgemm_nt_nosl(custream, d_stride_nt_lut, a_ptr, b_ptr, c_ptr, blk_size_, stride_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, stride_lut_heads, stride_nt_lut_dim);

    //     BlocksparseMaskedSoftmax_nosl<float,float2>(custream, d_stride_nn_lut, sm_mask1, c_ptr, y1_ptr, blk_size_, stride_blocks, batch_dim, heads_, ctx_blks_a_, stride_lut_heads, stride_nn_lut_dim, stride_nn_max, stride_mask_heads, scale);

    // }
    
    // for(int i=0;i<1000;i++){

    //     hipDeviceSynchronize();
    //     gettimeofday(&(GET_TIME_START1), NULL);
    //     gettimeofday(&(GET_TIME_START), NULL);
    
    //     bst_sgemm_nt_nosl(custream, d_stride_nt_lut, a_ptr, b_ptr, c_ptr, blk_size_, stride_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, stride_lut_heads, stride_nt_lut_dim);

    //     hipDeviceSynchronize();
    //     gettimeofday(&(GET_TIME_END), NULL);
    //     time1 = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;

    //     hipDeviceSynchronize();
    //     gettimeofday(&(GET_TIME_START), NULL);      

    //     BlocksparseMaskedSoftmax_nosl<float,float2>(custream, d_stride_nn_lut, sm_mask1, c_ptr, y1_ptr, blk_size_, stride_blocks, batch_dim, heads_, ctx_blks_a_, stride_lut_heads, stride_nn_lut_dim, stride_nn_max, stride_mask_heads, scale);

    //     hipDeviceSynchronize();
    //     gettimeofday(&(GET_TIME_END), NULL);
    //     gettimeofday(&(GET_TIME_END1), NULL);


    //     time2 = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;
    //     ntime = (GET_TIME_END1.tv_sec - GET_TIME_START1.tv_sec) * 1000.0 + (GET_TIME_END1.tv_usec - GET_TIME_START1.tv_usec) / 1000.0;

    //     ntime_avg+=ntime;
    //     ntime_min = ntime_min > ntime ? ntime : ntime_min;

    //     time_avg1+=time1;
    //     time_min1 = time_min1 > time1 ? time1 : time_min1;

    //     time_avg2+=time2;
    //     time_min2 = time_min2 > time2 ? time2 : time_min2;

    // }

    // printf("sddmm_time_avg:%f \n", time_avg1/1000.0);
    // printf("sddmm_time_min:%f \n", time_min1);

    // printf("sfmx_time_avg:%f \n", time_avg2/1000.0);
    // printf("sfmx_time_min:%f \n", time_min2);

    // printf("nofusion_time_avg:%f \n", ntime_avg/1000.0);
    // printf("nofusion_time_min:%f \n", ntime_min);

    return 0;

}