#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "validate.h"
#include "common.h"
#include "blocksparse.h"
#include "fusion_blocksparse.h"

int main(){

    const uint batch_dim = 32;
    const uint heads_ = 4;
    const uint head_state_ = 128;
    const uint blk_size_= 32;
    const uint seq_len = 1024;
    const uint state = 512;
    const uint m = batch_dim*seq_len;
    const uint n = state;

    const uint nt_op = 0;
    const uint nn_op = 1; 
    const uint tn_op = 2;
    int count = 0;
    const uint ctx_blks_a_ = seq_len/blk_size_;
    const uint ctx_blks_b_ = seq_len/blk_size_;
    const uint ctx_blks_c_ = seq_len/blk_size_;
    const float scale = 1.0/sqrt(head_state_);

    const uint local_lut_heads = 1;
    const uint local_mask_heads = 1;
    uint local_nn_max = 2;
    uint local_tn_max = 2;
    const uint local_attn_ctx = 32;

    uint local_nt_lut_dim = 0;
    uint local_nn_lut_dim = 0;
    uint local_tn_lut_dim = 0;

    uint *layout = (uint*)malloc(sizeof(uint)*ctx_blks_a_*ctx_blks_a_);

    int extra_diag = local_attn_ctx / blk_size_;

    for(int i=0;i<(ctx_blks_a_*ctx_blks_a_);i++){

        layout[i] = 1;
    }

     for(int i=0;i<ctx_blks_a_;i++){
        for(int j=0;j<ctx_blks_a_;j++){
            
            if((extra_diag!=0)&&((j+extra_diag)<i) || (j>i)){
                layout[i*ctx_blks_a_+j] = 0;
            }

        }
    }

    // for(int i=0;i<ctx_blks_a_;i++){
    //     for(int j=0;j<ctx_blks_a_;j++){

    //         printf("%u ", layout[i*ctx_blks_a_+j]);

    //     }

    //     printf("\n");
    // }

    // printf("\n");

    for(int i=0;i<(ctx_blks_a_*ctx_blks_a_);i++){

        if(layout[i] == 1){
            local_nt_lut_dim++;
        }

    }

    local_nn_lut_dim = local_nt_lut_dim+ctx_blks_a_;
    local_tn_lut_dim = local_nn_lut_dim;


    uint2 *local_nt_lut = (uint2*)malloc(sizeof(uint2)*(local_nt_lut_dim));

    for(int i=0;i<(ctx_blks_a_*ctx_blks_a_);i++){

        if(layout[i] == 1){
            local_nt_lut[count].x = i / ctx_blks_a_;
            local_nt_lut[count].y = i % ctx_blks_a_;
            count++;
        }

    }

    uint2 *d_local_nt_lut;
    hipMalloc((void**)&d_local_nt_lut, sizeof(uint2)*local_nt_lut_dim);
    hipMemcpy(d_local_nt_lut, local_nt_lut, sizeof(uint2)*local_nt_lut_dim, hipMemcpyHostToDevice);


    uint2 *local_nn_lut = (uint2*)malloc(sizeof(uint2)*local_nn_lut_dim);

    for(int i=0;i<local_nn_lut_dim;i++){

        local_nn_lut[i].x = 0;
        local_nn_lut[i].y = 0;
    }

    for(int i=0;i<local_nt_lut_dim;i++){
        
        local_nn_lut[local_nt_lut[i].x].y++;
        local_nn_lut[ctx_blks_a_+i].x = i;
        local_nn_lut[ctx_blks_a_+i].y = local_nt_lut[i].y;
        
    }

    local_nn_lut[0].x = ctx_blks_a_;

    for(int i=1;i<ctx_blks_a_;i++){

        local_nn_lut[i].x = local_nn_lut[i-1].x+local_nn_lut[i-1].y;
    }

    for(int i=0;i<ctx_blks_a_;i++){

        local_nn_max = local_nn_max > local_nn_lut[i].y ? local_nn_max : local_nn_lut[i].y;

    }

    // for(int i=0;i<local_nn_lut_dim;i++){
    //     printf("%u %u \n", local_nn_lut[i].x, local_nn_lut[i].y);
    // }

    uint2 *d_local_nn_lut;
    hipMalloc((void**)&d_local_nn_lut, sizeof(uint2)*local_nn_lut_dim);
    hipMemcpy(d_local_nn_lut, local_nn_lut, sizeof(uint2)*local_nn_lut_dim, hipMemcpyHostToDevice);


    uint local_blocks = local_nt_lut_dim;

    bool *mask = (bool*)malloc(sizeof(bool)*local_blocks*blk_size_*blk_size_);

    for(int i=0;i<(local_blocks*blk_size_*blk_size_);i++){

        mask[i] = 1;
    }

    for(int i=0;i<local_blocks;i++){

        uint i_idx = local_nt_lut[i].x;
        uint j_idx = local_nt_lut[i].y;

        for(int j=0;j<(blk_size_);j++){

            for(int l=0;l<blk_size_;l++){

                if(i_idx == j_idx){
                    if(l>j)
                        mask[i*blk_size_*blk_size_+j*blk_size_+l] = 0;
                }

                uint row_idx = (i_idx)*blk_size_+j;
                uint col_idx = (j_idx)*blk_size_+l;

                if((col_idx>row_idx) || ((col_idx+local_attn_ctx) <= row_idx)){
                    mask[i*(blk_size_*blk_size_)+j*blk_size_+l] = 0;
                }
            }
        }
    }

    unsigned int *local_mask_np = (unsigned int*)malloc(sizeof(unsigned int)*local_blocks*blk_size_);
    unsigned int *local_mask = (unsigned int*)malloc(sizeof(unsigned int)*local_blocks*blk_size_);

    for(int i=0;i<(local_blocks*blk_size_);i++){
        local_mask_np[i] = 0;
        for(int j=0;j<32;j++){
            bool keep = mask[i*blk_size_+j];
            unsigned int temp;
            temp = keep << (j);
            local_mask_np[i] = local_mask_np[i] | temp;  
        }
    }

    for(int i=0; i<local_blocks; i++){
        for(int j=0; j<blk_size_; j++){
            local_mask[j*local_blocks+i] = local_mask_np[i*blk_size_+j];
        }
    }

    unsigned int *sm_mask;
    hipMalloc((void**)&sm_mask, sizeof(unsigned int)*(local_blocks*blk_size_));
    hipMemcpy(sm_mask, local_mask_np, sizeof(unsigned int)*(local_blocks*blk_size_), hipMemcpyHostToDevice);

    unsigned *sm_mask1;
    hipMalloc((void**)&sm_mask1, sizeof(unsigned)*(local_blocks*blk_size_));
    hipMemcpy(sm_mask1, local_mask, sizeof(unsigned)*(local_blocks*blk_size_), hipMemcpyHostToDevice);

    float *a_ptr_cpu = (float*)malloc(sizeof(float)*m*n);
    float *b_ptr_cpu = (float*)malloc(sizeof(float)*m*n);

    // char q_file[] = "/home/songshuhui/Desktop/Transformer-0523/comparison_precision/q";
    // char k_file[] = "/home/songshuhui/Desktop/Transformer-0523/comparison_precision/k";
    // readbinary(q_file, a_ptr_cpu, m*n);  
    // readbinary(k_file, b_ptr_cpu, m*n);  

    for(int i=0;i<m*n;i++){

        a_ptr_cpu[i] = 1.0;
        b_ptr_cpu[i] = 1.0;

    }


    float *a_ptr, *b_ptr;
    hipMalloc((void**)&a_ptr, sizeof(float)*m*n);
    hipMalloc((void**)&b_ptr, sizeof(float)*m*n);

    hipMemcpy(a_ptr, a_ptr_cpu, sizeof(float)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(b_ptr, b_ptr_cpu, sizeof(float)*m*n, hipMemcpyHostToDevice);

    

    uint2 *rblk_lutc = (uint2*)malloc(sizeof(uint2)*(local_blocks));
    count = 0;
    for(int i=0;i<ctx_blks_a_;i++){
        for(int j=0;j<local_nn_lut[i].y;j++){
            rblk_lutc[count].x = count;
            rblk_lutc[count].y = i;
            count++;
        }
    }

    
    uint2 *rblk_lut;
    hipMalloc((void**)&rblk_lut, sizeof(uint2)*local_blocks);
    hipMemcpy(rblk_lut, rblk_lutc, sizeof(uint2)*(local_blocks), hipMemcpyHostToDevice);

    float *y_ptr;
    hipMalloc((void**)&y_ptr, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_*blk_size_));

    // int *rblk_flag_c = (int*)malloc(sizeof(int)*ctx_blks_a_*batch_dim*heads_);
    // for(int i=0;i<(batch_dim*heads_);i++){
    //     for(int j=0;j<ctx_blks_a_;j++){
    //         rblk_flag_c[i*ctx_blks_a_+j] = local_nn_lut[ctx_blks_a_-j-1].y;
    //     }
    // }

    int *rblk_flag;
    hipMalloc((void**)&rblk_flag, sizeof(int)*ctx_blks_a_*batch_dim*heads_);
    // hipMemcpy(rblk_flag, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);

    int *rblk_flag1;
    hipMalloc((void**)&rblk_flag1, sizeof(int)*ctx_blks_a_*batch_dim*heads_);
    // hipMemcpy(rblk_flag1, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);

    float *Max;
    hipMalloc((void**)&Max, sizeof(float)*batch_dim*heads_*local_blocks*blk_size_);

    float *Sum;
    hipMalloc((void**)&Sum, sizeof(float)*batch_dim*heads_*local_blocks*blk_size_);

    float *Maxc = (float*)malloc(sizeof(float)*batch_dim*heads_*local_blocks*blk_size_);
    float *Sumc = (float*)malloc(sizeof(float)*batch_dim*heads_*local_blocks*blk_size_);
    float *yc = (float*)malloc(sizeof(float)*blk_size_*blk_size_*batch_dim*heads_*local_blocks); 
    float *yc1 = (float*)malloc(sizeof(float)*blk_size_*blk_size_*batch_dim*heads_*local_blocks); 

    bhalf *c_ptr;
    hipMalloc((void**)&c_ptr, sizeof(bhalf)*(batch_dim*heads_*local_blocks*blk_size_*blk_size_));
    bhalf *c_ptr_cpu = (bhalf*)malloc(sizeof(bhalf)*(batch_dim*heads_*local_blocks*blk_size_*blk_size_));

    float *y1_ptr;
    hipMalloc((void**)&y1_ptr, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_*blk_size_));

    double time=0, ntime=0, time1=0, time2=0;
    double time_avg = 0, ntime_avg = 0; 
    double time_min = 1000, ntime_min = 1000;


    double time_avg1 = 0, time_avg2 = 0;
    double time_min1 = 1000, time_min2 = 1000;

    struct timeval GET_TIME_START, GET_TIME_END, GET_TIME_START1, GET_TIME_END1;

    hipStream_t custream;
    hipStreamCreate(&custream); 

    for(int i=0;i<1000;i++){
        
        hipDeviceSynchronize();
        fusion_attention_local<float, float2>(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, d_local_nn_lut, rblk_flag, rblk_flag1, sm_mask, Max, Sum, y_ptr,
                        blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                        local_lut_heads, local_nt_lut_dim, local_mask_heads, scale);

        hipDeviceSynchronize();
    
        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));

    }
    int loops = 1000;

    for(int i=0;i<loops;i++){
        
        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START), NULL);

        fusion_attention_local<float, float2>(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, d_local_nn_lut, rblk_flag, rblk_flag1, sm_mask, Max, Sum, y_ptr,
                        blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                        local_lut_heads, local_nt_lut_dim, local_mask_heads, scale);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        double time = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;

        time_avg+=time;
        time_min = time_min > time ? time : time_min;

        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));

    }
    
    printf("local_time_avg:%f \n", time_avg/loops);
    printf("local_time_min:%f \n", time_min);


    // hipMemcpy(yc1, y_ptr, sizeof(float)*(local_blocks*blk_size_*blk_size_), hipMemcpyDeviceToHost);

    // for(int i=0;i<1;i++){
    //     for(int j=0;j<blk_size_;j++){
    //         for(int l=0;l<blk_size_;l++){
    //             printf("%f ", yc1[i*blk_size_*blk_size_+j*blk_size_+l]);
    //         }
    //         printf("\n");
    //     }

    //     printf("\n");

    // }

    // return 0;

    time_avg = 0;
    time_min = 1000;

    for(int i=0;i<loops;i++){

        fusion_attention1<float, float2>(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, d_local_nn_lut, rblk_flag, rblk_flag1, rblk_lut,sm_mask, Max, Sum, y_ptr,
                    blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                    local_lut_heads, local_nt_lut_dim, local_mask_heads, scale);

        // hipMemcpy(rblk_flag, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);
        // hipMemcpy(rblk_flag1, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);
        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));


    }




    for(int i=0;i<loops;i++){

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START), NULL);

        fusion_attention1<float, float2>(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, d_local_nn_lut, rblk_flag, rblk_flag1, rblk_lut, sm_mask, Max, Sum, y_ptr,
                        blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                        local_lut_heads, local_nt_lut_dim, local_mask_heads, scale);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        double time = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;

        time_avg+=time;
        time_min = time_min > time ? time : time_min;

        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));

    }

    printf("fusion1_time_avg:%f \n", time_avg/loops);
    printf("fusion1_time_min:%f \n", time_min);
    

    time_avg = 0;
    time_min = 1000;


    for(int i=0;i<loops;i++){

        fusion_attention2<float, float2>(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, d_local_nn_lut, rblk_flag, rblk_flag1, sm_mask, Max, Sum, y1_ptr,
                    blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                    local_lut_heads, local_nt_lut_dim, local_mask_heads, scale);

        // hipMemcpy(rblk_flag, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);
        // hipMemcpy(rblk_flag1, rblk_flag_c, sizeof(int)*(ctx_blks_a_*batch_dim*heads_), hipMemcpyHostToDevice);
        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));


    }


    for(int i=0;i<loops;i++){

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START), NULL);

        fusion_attention2<float, float2>(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, d_local_nn_lut, rblk_flag, rblk_flag1, sm_mask, Max, Sum, y1_ptr,
                        blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, 
                        local_lut_heads, local_nt_lut_dim, local_mask_heads, scale);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        double time = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;

        time_avg+=time;
        time_min = time_min > time ? time : time_min;

        hipMemset(rblk_flag, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(rblk_flag1, 0, sizeof(int)*(ctx_blks_a_*batch_dim*heads_));
        hipMemset(Max, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));
        hipMemset(Sum, 0, sizeof(float)*(batch_dim*heads_*local_blocks*blk_size_));

    }

    printf("fusion2_time_avg:%f \n", time_avg/loops);
    printf("fusion2_time_min:%f \n", time_min);
  
    // hipMemcpy(yc, y1_ptr, sizeof(float)*(local_blocks*blk_size_*blk_size_), hipMemcpyDeviceToHost);

    // for(int i=0;i<1;i++){
    //     for(int j=0;j<blk_size_;j++){
    //         for(int l=0;l<blk_size_;l++){
    //             printf("%f %f\n", yc[i*blk_size_*blk_size_+j*blk_size_+l], yc1[i*blk_size_*blk_size_+j*blk_size_+l]);
    //         }
    //         printf("\n");
    //     }

    //     printf("\n");

    // }
    // return 0;

    for(int i=0;i<1000;i++){

        bst_sgemm_nt(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, local_lut_heads, local_nt_lut_dim);

        BlocksparseMaskedSoftmax<float,float2>(custream, d_local_nn_lut, sm_mask1, c_ptr, y1_ptr, blk_size_, local_blocks, batch_dim, heads_, ctx_blks_a_, local_lut_heads, local_nn_lut_dim, local_nn_max, local_mask_heads, scale);

    }
    
    for(int i=0;i<1000;i++){

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START1), NULL);
        gettimeofday(&(GET_TIME_START), NULL);
    
        bst_sgemm_nt(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, local_lut_heads, local_nt_lut_dim);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        time1 = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_START), NULL);      

        BlocksparseMaskedSoftmax<float,float2>(custream, d_local_nn_lut, sm_mask1, c_ptr, y1_ptr, blk_size_, local_blocks, batch_dim, heads_, ctx_blks_a_, local_lut_heads, local_nn_lut_dim, local_nn_max, local_mask_heads, scale);

        hipDeviceSynchronize();
        gettimeofday(&(GET_TIME_END), NULL);
        gettimeofday(&(GET_TIME_END1), NULL);


        time2 = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;
        ntime = (GET_TIME_END1.tv_sec - GET_TIME_START1.tv_sec) * 1000.0 + (GET_TIME_END1.tv_usec - GET_TIME_START1.tv_usec) / 1000.0;

        ntime_avg+=ntime;
        ntime_min = ntime_min > ntime ? ntime : ntime_min;

        time_avg1+=time1;
        time_min1 = time_min1 > time1 ? time1 : time_min1;

        time_avg2+=time2;
        time_min2 = time_min2 > time2 ? time2 : time_min2;

    }

    printf("sddmm_time_avg:%f \n", time_avg1/1000.0);
    printf("sddmm_time_min:%f \n", time_min1);

    printf("sfmx_time_avg:%f \n", time_avg2/1000.0);
    printf("sfmx_time_min:%f \n", time_min2);

    printf("nofusion_time_avg:%f \n", ntime_avg/1000.0);
    printf("nofusion_time_min:%f \n", ntime_min);

    // ntime_avg = 0;
    // ntime_min = 1000;

    // time_avg1 = 0;
    // time_min1 = 1000;

    // time_avg2 = 0;
    // time_min2 = 1000;

    // for(int i=0;i<1000;i++){

    //     bst_sgemm_nt_nosl(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, local_lut_heads, local_nt_lut_dim);

    //     BlocksparseMaskedSoftmax_nosl<float,float2>(custream, d_local_nn_lut, sm_mask1, c_ptr, y1_ptr, blk_size_, local_blocks, batch_dim, heads_, ctx_blks_a_, local_lut_heads, local_nn_lut_dim, local_nn_max, local_mask_heads, scale);

    // }
    
    // for(int i=0;i<1000;i++){

    //     hipDeviceSynchronize();
    //     gettimeofday(&(GET_TIME_START1), NULL);
    //     gettimeofday(&(GET_TIME_START), NULL);
    
    //     bst_sgemm_nt_nosl(custream, d_local_nt_lut, a_ptr, b_ptr, c_ptr, blk_size_, local_blocks, batch_dim, ctx_blks_a_, ctx_blks_b_, heads_, head_state_, local_lut_heads, local_nt_lut_dim);

    //     hipDeviceSynchronize();
    //     gettimeofday(&(GET_TIME_END), NULL);
    //     time1 = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;

    //     hipDeviceSynchronize();
    //     gettimeofday(&(GET_TIME_START), NULL);      

    //     BlocksparseMaskedSoftmax_nosl<float,float2>(custream, d_local_nn_lut, sm_mask1, c_ptr, y1_ptr, blk_size_, local_blocks, batch_dim, heads_, ctx_blks_a_, local_lut_heads, local_nn_lut_dim, local_nn_max, local_mask_heads, scale);

    //     hipDeviceSynchronize();
    //     gettimeofday(&(GET_TIME_END), NULL);
    //     gettimeofday(&(GET_TIME_END1), NULL);


    //     time2 = (GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0;
    //     ntime = (GET_TIME_END1.tv_sec - GET_TIME_START1.tv_sec) * 1000.0 + (GET_TIME_END1.tv_usec - GET_TIME_START1.tv_usec) / 1000.0;

    //     ntime_avg+=ntime;
    //     ntime_min = ntime_min > ntime ? ntime : ntime_min;

    //     time_avg1+=time1;
    //     time_min1 = time_min1 > time1 ? time1 : time_min1;

    //     time_avg2+=time2;
    //     time_min2 = time_min2 > time2 ? time2 : time_min2;

    // }

    // printf("sddmm_time_avg:%f \n", time_avg1/1000.0);
    // printf("sddmm_time_min:%f \n", time_min1);

    // printf("sfmx_time_avg:%f \n", time_avg2/1000.0);
    // printf("sfmx_time_min:%f \n", time_min2);

    // printf("nofusion_time_avg:%f \n", ntime_avg/1000.0);
    // printf("nofusion_time_min:%f \n", ntime_min);

    return 0;


}