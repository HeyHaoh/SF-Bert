#include "hip/hip_runtime.h"
// #define TRANSFORMER_DEBUG
#define DRAW_MAT

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <sys/time.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "cuda_kernels.h"
#include "validate.h"
#include "common.h"
#include "embedding.h"
#include "layernorm.h"
#include "matmul.h"
#include "ewops.h"
#include "optimizer.h"

struct timeval GET_TIME_START, GET_TIME_END;

using namespace std;

void time_check_begin()
{
    gettimeofday(&(GET_TIME_START), NULL);
}

double time_check_end()
{   
    gettimeofday(&(GET_TIME_END), NULL);
    return ((GET_TIME_END.tv_sec - GET_TIME_START.tv_sec) * 1000.0 + (GET_TIME_END.tv_usec - GET_TIME_START.tv_usec) / 1000.0);
}



int main(int argc, char* argv[]){

    if(argc != 6)
    {
        printf("./transformer_fp32 batch_size num_layers seq_len head_num size_per_head\n");
        printf("e.g., ./transformer_fp32 1 12 128 12 64\n");
        return 0;
    }

    const int batch_size = atoi(argv[1]);
    const int num_layers = atoi(argv[2]);
    const int seq_len = atoi(argv[3]);
    const int head_num = atoi(argv[4]);
    const int size_per_head = atoi(argv[5]);
    const int epoch = 0; 
    const int vocab_size = 256;
    const int state = head_num * size_per_head;
    const int m = batch_size * seq_len;
    const int k = state;
    const int n = state;
    float scaler = 1.0 / (sqrt(size_per_head));
    float grad_sum_sum = 0.0;
    float clip_norm = 1.0;
    float norm_scale = 0.0;
    const float probe = 1.0-0.05;
    const float beta1 = 0.9;
    const float beta2 = 0.999;
    const float learning_rate = 0.0005;
    const float epsilon = 0.00000001;
    float *loss = (float*)malloc(sizeof(float));

    float global_norm = 0;
    float grad_scale = 1.0;
    float clip_sigma = 0;

    float lr  = 0.0;
    int global_step = 1;
    float beta1_power = 1.0;
    float beta2_power = 1.0;

    const float alpha = 1.0f;
    const float beta  = 0.0f;

    int offset = 0;
    int offset1 = 0;
    double used_time = 0.0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    srand((unsigned int)time(NULL));

    float *q_w = (float*)malloc(sizeof(float) * n * n * num_layers);
    float *k_w = (float*)malloc(sizeof(float) * n * n * num_layers);
    float *v_w = (float*)malloc(sizeof(float) * n * n * num_layers);
    float *a_w = (float*)malloc(sizeof(float) * n * n * num_layers);
    float *m1_w = (float*)malloc(sizeof(float) * n * n * 4 * num_layers);
    float *m2_w = (float*)malloc(sizeof(float) * n * n * 4 * num_layers);

    for(int i=0;i<num_layers;i++){

        random_normalInit(q_w+(i*n*n), 0, 0.02, n*n);
        random_normalInit(k_w+(i*n*n), 0, 0.02, n*n);
        random_normalInit(v_w+(i*n*n), 0, 0.02, n*n);
        random_normalInit(a_w+(i*n*n), 0, 0.02/num_layers, n*n);
        random_normalInit(m1_w+(i*n*n*4), 0, 0.02, n*n*4);
        random_normalInit(m2_w+(i*n*n*4), 0, 0.02, n*n*4);
    }

    float *d_qw, *d_kw, *d_vw, *d_aw, *d_m1w, *d_m2w;
    hipMalloc((void **)&d_qw, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_kw, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_vw, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_aw, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_m1w, sizeof(float)*n*n*4*num_layers);
    hipMalloc((void **)&d_m2w, sizeof(float)*n*n*4*num_layers);
    
    float *q_b = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(q_b, 0.0, n*num_layers);
    float *k_b = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(k_b, 0.0, n*num_layers);
    float *v_b = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(v_b, 0.0, n*num_layers);
    float *a_b = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(a_b, 0.0, n*num_layers);
    float *m1_b = (float*)malloc(sizeof(float) * n * 4 * num_layers);
    constantInit(m1_b, 0.0, n*4*num_layers);
    float *m2_b = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(m2_b, 0.0, n*num_layers);
    float *norm_a_g = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(norm_a_g, 1.0, n*num_layers);
    float *norm_a_b = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(norm_a_b, 0.0, n*num_layers);
    float *norm_m_g = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(norm_m_g, 1.0, n*num_layers);
    float *norm_m_b = (float*)malloc(sizeof(float) * n * num_layers);
    constantInit(norm_m_b, 0.0, n*num_layers);
    float *x_embed = (float*)malloc(sizeof(float) * vocab_size * state);
    random_normalInit(x_embed, 0, 0.02, vocab_size * state);
    float *p_embed = (float*)malloc(sizeof(float) * seq_len * state);
    random_normalInit(p_embed, 0, 0.01, seq_len * state);

    float *d_qb, *d_kb, *d_vb, *d_ab, *d_m1b, *d_m2b, *d_norm_ag, *d_norm_ab, *d_norm_mg, *d_norm_mb, *d_xembed, *d_pembed;
    hipMalloc((void **)&d_qb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_kb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_vb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_ab, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_m1b, sizeof(float)*n*4*num_layers);
    hipMalloc((void **)&d_m2b, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_ag, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_ab, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_mg, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_mb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_xembed, sizeof(float)*vocab_size*state);
    hipMalloc((void **)&d_pembed, sizeof(float)*seq_len*state);

    hipMemcpy(d_qw, q_w, sizeof(float)*n*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_kw, k_w, sizeof(float)*n*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_vw, v_w, sizeof(float)*n*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_aw, a_w, sizeof(float)*n*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_m1w, m1_w, sizeof(float)*n*n*4*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_m2w, m2_w, sizeof(float)*n*n*4*num_layers, hipMemcpyHostToDevice);

    hipMemcpy(d_qb, q_b, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_kb, k_b, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_vb, v_b, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_ab, a_b, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_m1b, m1_b, sizeof(float)*n*4*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_m2b, m2_b, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_norm_ag, norm_a_g, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_norm_ab, norm_a_b, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_norm_mg, norm_m_g, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);
    hipMemcpy(d_norm_mb, norm_m_b, sizeof(float)*n*num_layers, hipMemcpyHostToDevice);

    hipMemcpy(d_xembed, x_embed, sizeof(float)*vocab_size*state, hipMemcpyHostToDevice);
    hipMemcpy(d_pembed, p_embed, sizeof(float)*seq_len*state, hipMemcpyHostToDevice);



    float *embed_lookup, *x_after_dropout, *p_after_dropout;
    hipMalloc((void **)&embed_lookup, sizeof(float)*m*n);
    hipMalloc((void **)&x_after_dropout, sizeof(float)*m*n);
    hipMalloc((void **)&p_after_dropout, sizeof(float)*seq_len*n);

    float *embed_add;
    hipMalloc((void **)&embed_add, sizeof(float)*m*n*(num_layers+1));

    float *norm_a, *norm_a_mean, *norm_a_rstd;
    hipMalloc((void **)&norm_a, sizeof(float)*m*n*num_layers);
    hipMalloc((void **)&norm_a_mean, sizeof(float)*m*num_layers);
    hipMalloc((void **)&norm_a_rstd, sizeof(float)*m*num_layers);

    float *q_out, *k_out, *v_out;
    hipMalloc((void **)&q_out, sizeof(float)*m*n);
    hipMalloc((void **)&k_out, sizeof(float)*m*n);
    hipMalloc((void **)&v_out, sizeof(float)*m*n);

    float *q_bias_out, *k_bias_out, *v_bias_out;
    hipMalloc((void **)&q_bias_out, sizeof(float)*m*n);
    hipMalloc((void **)&k_bias_out, sizeof(float)*m*n);
    hipMalloc((void **)&v_bias_out, sizeof(float)*m*n);

    float *q_reshape, *k_reshape, *v_reshape;
    hipMalloc((void **)&q_reshape, sizeof(float)*m*n*num_layers);
    hipMalloc((void **)&k_reshape, sizeof(float)*m*n*num_layers);
    hipMalloc((void **)&v_reshape, sizeof(float)*m*n*num_layers);

    float *qk_out, *softmax_out, *sv_out, *sv_out_reshape;
    hipMalloc((void **)&qk_out, sizeof(float)*batch_size*head_num*seq_len*seq_len);
    hipMalloc((void **)&softmax_out, sizeof(float)*batch_size*head_num*seq_len*seq_len*num_layers);
    hipMalloc((void **)&sv_out, sizeof(float)*m*n);
    hipMalloc((void **)&sv_out_reshape, sizeof(float)*m*n*num_layers);

    float *a_out, *a_bias_out;
    hipMalloc((void **)&a_out, sizeof(float)*m*n);
    hipMalloc((void **)&a_bias_out, sizeof(float)*m*n);

    float *a_after_dropout, *add_1;
    hipMalloc((void **)&a_after_dropout, sizeof(float)*m*n);
    hipMalloc((void **)&add_1, sizeof(float)*m*n*num_layers);

    float *norm_m, *norm_m_mean, *norm_m_rstd;
    hipMalloc((void **)&norm_m, sizeof(float)*m*n*num_layers);
    hipMalloc((void **)&norm_m_mean, sizeof(float)*m*num_layers);
    hipMalloc((void **)&norm_m_rstd, sizeof(float)*m*num_layers);

    float *m1_out, *m1_gelu_out, *m1_bias_out;
    hipMalloc((void **)&m1_out, sizeof(float)*m*n*4);
    hipMalloc((void **)&m1_gelu_out, sizeof(float)*m*n*4*num_layers);
    hipMalloc((void **)&m1_bias_out, sizeof(float)*m*n*4*num_layers);

    float *m2_out, *m2_bias_out;
    hipMalloc((void **)&m2_out, sizeof(float)*m*n);
    hipMalloc((void **)&m2_bias_out, sizeof(float)*m*n);

    float *m_after_dropout;
    hipMalloc((void **)&m_after_dropout, sizeof(float)*m*n);

    float *logits, *softmax_logits;
    hipMalloc((void **)&logits, sizeof(float)*m*vocab_size);
    hipMalloc((void **)&softmax_logits, sizeof(float)*m*vocab_size);

    float *d_qdw, *d_kdw, *d_vdw, *d_adw, *d_m1dw, *d_m2dw;
    hipMalloc((void **)&d_qdw, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_kdw, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_vdw, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_adw, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_m1dw, sizeof(float)*n*n*4*num_layers);
    hipMalloc((void **)&d_m2dw, sizeof(float)*n*n*4*num_layers);

    float *d_qdb, *d_kdb, *d_vdb, *d_adb, *d_m1db, *d_m2db, *d_norm_adg, *d_norm_adb, *d_norm_mdg, *d_norm_mdb;
    hipMalloc((void **)&d_qdb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_kdb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_vdb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_adb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_m1db, sizeof(float)*n*num_layers*4);
    hipMalloc((void **)&d_m2db, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_adg, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_adb, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_mdg, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_mdb, sizeof(float)*n*num_layers);

    float *logits_dx, *logits_dw;
    hipMalloc((void **)&logits_dx, sizeof(float)*(num_layers+1)*m*n);
    hipMalloc((void **)&logits_dw, sizeof(float)*n*vocab_size);

    float *entropy_grad = (float*)malloc(sizeof(float)*m*vocab_size);
    hipMalloc((void **)&entropy_grad, sizeof(float)*m*vocab_size);

    float *dx1, *dx2, *dx3, *grad_add1, *dx_41, *dx_42;
    hipMalloc((void **)&dx1, sizeof(float)*n*m);
    hipMalloc((void **)&dx2, sizeof(float)*n*m);
    hipMalloc((void **)&dx3, sizeof(float)*n*m);
    hipMalloc((void **)&grad_add1, sizeof(float)*n*m);
    hipMalloc((void **)&dx_41, sizeof(float)*n*m*4);
    hipMalloc((void **)&dx_42, sizeof(float)*n*m*4);

    float *q_grad, *k_grad, *v_grad, *q_grad_trans;
    hipMalloc((void **)&q_grad, sizeof(float)*n*m);
    hipMalloc((void **)&q_grad_trans, sizeof(float)*n*m);
    hipMalloc((void **)&k_grad, sizeof(float)*n*m);
    hipMalloc((void **)&v_grad, sizeof(float)*n*m);

    float *q_grad_reshape, *k_grad_reshape, *v_grad_reshape;
    hipMalloc((void **)&q_grad_reshape, sizeof(float)*n*m);
    hipMalloc((void **)&k_grad_reshape, sizeof(float)*n*m);
    hipMalloc((void **)&v_grad_reshape, sizeof(float)*n*m);

    float *sv_grad, *softmaxgrad;
    hipMalloc((void **)&sv_grad, sizeof(float)*batch_size*head_num*seq_len*seq_len);
    hipMalloc((void **)&softmaxgrad, sizeof(float)*batch_size*head_num*seq_len*seq_len);

    float *p_embed_grad, *p_embed_dw, *embed_grad, *logits_dw_trans, *embed_add_out;
    hipMalloc((void**)&p_embed_grad, sizeof(float)*seq_len*state);
    hipMalloc((void**)&p_embed_dw, sizeof(float)*seq_len*state);
    hipMalloc((void**)&embed_grad, sizeof(float)*vocab_size*state);
    hipMalloc((void**)&logits_dw_trans, sizeof(float)*vocab_size*state);
    hipMalloc((void**)&embed_add_out, sizeof(float)*vocab_size*state);

    
    
    char xs_path[100];
    char temp[100];
    int *xs_int = (int*)malloc(sizeof(int) * m);
    unsigned char *xs_char = (unsigned char*)malloc(sizeof(unsigned char) * m);
    
    char ys_path[100];
    char y_temp[100];
    int *ys_int = (int*)malloc(sizeof(int) * m);
    unsigned char *ys_char = (unsigned char*)malloc(sizeof(unsigned char) * m);
    int *one_hot_ys = (int*)malloc(sizeof(int)*m*vocab_size);

    float *gradients_sum;
    hipMalloc((void**)&gradients_sum, sizeof(float)*(16*num_layers+2));
    hipMemset(gradients_sum, 0.0, sizeof(float)*(16*num_layers+2));

    float *gradients_sumc = (float*)malloc(sizeof(float)*num_layers*16+2);
    for(int i=0;i<num_layers*16+2;i++){
        gradients_sumc[i] = 0.0;
    }

    

    float *d_norm_a_g_mt, *d_norm_a_g_vt, *d_norm_a_b_mt, *d_norm_a_b_vt;
    hipMalloc((void **)&d_norm_a_g_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_a_g_vt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_a_b_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_a_b_vt, sizeof(float)*n*num_layers);

    hipMemset(d_norm_a_g_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_norm_a_g_vt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_norm_a_b_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_norm_a_b_vt, 0.0, sizeof(float)*n*num_layers);

    float *d_norm_m_g_mt, *d_norm_m_g_vt, *d_norm_m_b_mt, *d_norm_m_b_vt;
    hipMalloc((void **)&d_norm_m_g_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_m_g_vt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_m_b_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_norm_m_b_vt, sizeof(float)*n*num_layers);

    hipMemset(d_norm_m_g_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_norm_m_g_vt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_norm_m_b_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_norm_m_b_vt, 0.0, sizeof(float)*n*num_layers);

    float *d_q_b_mt, *d_q_b_vt, *d_k_b_mt, *d_k_b_vt, *d_v_b_mt, *d_v_b_vt;
    hipMalloc((void **)&d_q_b_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_q_b_vt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_k_b_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_k_b_vt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_v_b_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_v_b_vt, sizeof(float)*n*num_layers);

    hipMemset(d_q_b_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_q_b_vt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_k_b_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_k_b_vt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_v_b_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_v_b_vt, 0.0, sizeof(float)*n*num_layers);

    float *d_q_w_mt, *d_q_w_vt, *d_k_w_mt, *d_k_w_vt, *d_v_w_mt, *d_v_w_vt;
    hipMalloc((void **)&d_q_w_mt, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_q_w_vt, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_k_w_mt, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_k_w_vt, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_v_w_mt, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_v_w_vt, sizeof(float)*n*n*num_layers);

    hipMemset(d_q_w_mt, 0.0, sizeof(float)*n*n*num_layers);
    hipMemset(d_q_w_vt, 0.0, sizeof(float)*n*n*num_layers);
    hipMemset(d_k_w_mt, 0.0, sizeof(float)*n*n*num_layers);
    hipMemset(d_k_w_vt, 0.0, sizeof(float)*n*n*num_layers);
    hipMemset(d_v_w_mt, 0.0, sizeof(float)*n*n*num_layers);
    hipMemset(d_v_w_vt, 0.0, sizeof(float)*n*n*num_layers);

    float *d_a_b_mt, *d_a_b_vt, *d_a_w_mt, *d_a_w_vt;
    hipMalloc((void **)&d_a_b_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_a_b_vt, sizeof(float)*n*num_layers);

    hipMemset(d_a_b_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_a_b_vt, 0.0, sizeof(float)*n*num_layers);

    hipMalloc((void **)&d_a_w_mt, sizeof(float)*n*n*num_layers);
    hipMalloc((void **)&d_a_w_vt, sizeof(float)*n*n*num_layers);

    hipMemset(d_a_w_mt, 0.0, sizeof(float)*n*n*num_layers);
    hipMemset(d_a_w_vt, 0.0, sizeof(float)*n*n*num_layers);

    float *d_m1_b_mt, *d_m1_b_vt;
    hipMalloc((void **)&d_m1_b_mt, sizeof(float)*n*4*num_layers);
    hipMalloc((void **)&d_m1_b_vt, sizeof(float)*n*4*num_layers);

    hipMemset(d_m1_b_mt, 0.0, sizeof(float)*n*4*num_layers);
    hipMemset(d_m1_b_vt, 0.0, sizeof(float)*n*4*num_layers);

    float *d_m1_w_mt, *d_m1_w_vt, *d_m2_w_mt, *d_m2_w_vt;
    hipMalloc((void **)&d_m1_w_mt, sizeof(float)*n*n*4*num_layers);
    hipMalloc((void **)&d_m1_w_vt, sizeof(float)*n*n*4*num_layers);
    hipMalloc((void **)&d_m2_w_mt, sizeof(float)*n*n*4*num_layers);
    hipMalloc((void **)&d_m2_w_vt, sizeof(float)*n*n*4*num_layers);

    hipMemset(d_m1_w_mt, 0.0, sizeof(float)*n*n*4*num_layers);
    hipMemset(d_m1_w_vt, 0.0, sizeof(float)*n*n*4*num_layers);
    hipMemset(d_m2_w_mt, 0.0, sizeof(float)*n*n*4*num_layers);
    hipMemset(d_m2_w_vt, 0.0, sizeof(float)*n*n*4*num_layers);

    float *d_m2_b_mt, *d_m2_b_vt;
    hipMalloc((void **)&d_m2_b_mt, sizeof(float)*n*num_layers);
    hipMalloc((void **)&d_m2_b_vt, sizeof(float)*n*num_layers);

    hipMemset(d_m2_b_mt, 0.0, sizeof(float)*n*num_layers);
    hipMemset(d_m2_b_vt, 0.0, sizeof(float)*n*num_layers);

    float *d_p_embed_mt, *d_p_embed_vt;
    hipMalloc((void **)&d_p_embed_mt, sizeof(float)*seq_len*state);
    hipMalloc((void **)&d_p_embed_vt, sizeof(float)*seq_len*state);

    hipMemset(d_p_embed_mt, 0.0, sizeof(float)*seq_len*state);
    hipMemset(d_p_embed_vt, 0.0, sizeof(float)*seq_len*state);

    float *d_x_embed_mt, *d_x_embed_vt;
    hipMalloc((void **)&d_x_embed_mt, sizeof(float)*vocab_size*state);
    hipMalloc((void **)&d_x_embed_vt, sizeof(float)*vocab_size*state);

    hipMemset(d_x_embed_mt, 0.0, sizeof(float)*vocab_size*state);
    hipMemset(d_x_embed_vt, 0.0, sizeof(float)*vocab_size*state);

    

    int *entropy_random = (int*)malloc(sizeof(int)*batch_size*seq_len*state*3);

    for(int i=0;i<batch_size*seq_len*state*3;i++){
        entropy_random[i] = rand();
    }

    unsigned int *d_xmask, *d_pmask;
    hipMalloc((void **)&d_xmask, sizeof(unsigned int)*seq_len*state);
    hipMalloc((void **)&d_pmask, sizeof(unsigned int)*5120);

    
    
    unsigned int *d_amask, *d_mmask;
    hipMalloc((void **)&d_amask, sizeof(unsigned int)*seq_len*state*num_layers);
    hipMalloc((void **)&d_mmask, sizeof(unsigned int)*seq_len*state*num_layers);

#ifdef DRAW_MAT

    float *qk_out_cpu = (float*)malloc(sizeof(float)*batch_size*head_num*seq_len*seq_len);
    float *softmax_out_cpu = (float*)malloc(sizeof(float)*batch_size*head_num*seq_len*seq_len);
    int *mask = (int*)malloc(sizeof(int)*seq_len*seq_len);
    int *d_mask;
    hipMalloc((void**)&d_mask, sizeof(int)*seq_len*seq_len);

#endif

    for(int iter=0;iter<1;iter++)
    {

        char xs_path_1[] = "xs";
        strcpy(xs_path, xs_path_1);
        sprintf(temp, "%d", 0);
        strcat(xs_path, temp);
        readbinary_char(xs_path, xs_char, m);
        for(int i=0;i<m;i++){
            xs_int[i] = (int)xs_char[i];  
        }

        char ys_path_1[] = "ys";
        strcpy(ys_path, ys_path_1);
        sprintf(y_temp, "%d", 0);
        strcat(ys_path, y_temp);
        readbinary_char(ys_path, ys_char, m);

       

        for(int i=0;i<m;i++){
            ys_int[i] = (int)ys_char[i];
        }
        to_one_hot(one_hot_ys, ys_int, batch_size, seq_len, vocab_size);

        int *d_xlabel, *d_ylabel;
        hipMalloc((void **)&d_xlabel, sizeof(int)*m);
        hipMalloc((void **)&d_ylabel, sizeof(int)*m*vocab_size);
        hipMemcpy(d_xlabel, xs_int, sizeof(int)*batch_size*seq_len, hipMemcpyHostToDevice);
        hipMemcpy(d_ylabel, one_hot_ys, sizeof(int)*m*vocab_size, hipMemcpyHostToDevice);

        float *d_loss;
        hipMalloc((void**)&d_loss, sizeof(float)*m);

         
#ifdef TRANSFORMER_DEBUG
         hipDeviceSynchronize();
         time_check_begin();
#endif

        embedding_lookup_gpu<<<m, n>>>(embed_lookup, d_xembed, d_xlabel, batch_size, seq_len, state, vocab_size);
#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("1 :embedding time: %f ms\n", used_time);
#endif


        int *d_entropy;
        hipMalloc((void **)&d_entropy, sizeof(int)*batch_size*seq_len*state*3);
        hipMemcpy(d_entropy, entropy_random, sizeof(int)*batch_size*seq_len*state*3, hipMemcpyHostToDevice);

#ifdef TRANSFORMER_DEBUG
         hipDeviceSynchronize();
         time_check_begin();
#endif

        GendropoutMask<<<m, n>>>(d_entropy, d_xmask, probe, seq_len*state);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("2 :gen x mask time: %f ms\n", used_time);
#endif

#ifdef TRANSFORMER_DEBUG
         hipDeviceSynchronize();
         time_check_begin();
#endif

        ApplydropoutMask<<<320, 512>>>(x_after_dropout, embed_lookup, d_xmask, probe, seq_len*state);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("3 :x apply dropout time: %f ms\n", used_time);
#endif

#ifdef TRANSFORMER_DEBUG
         hipDeviceSynchronize();
         time_check_begin();
#endif

        GendropoutMask<<<seq_len, n>>>(d_entropy, d_pmask, probe, seq_len*state);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("4 :gen p mask: %f ms\n", used_time);
#endif

#ifdef TRANSFORMER_DEBUG
         hipDeviceSynchronize();
         time_check_begin();
#endif

        ApplydropoutMask_SmallSize<<<10, 512>>>(p_after_dropout, d_pembed, d_pmask, probe, 5120);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("5 :p apply dropout time: %f ms\n", used_time);
#endif


#ifdef TRANSFORMER_DEBUG
         hipDeviceSynchronize();
         time_check_begin();
#endif
        tensor_add_matrix_gpu<<<m, n>>>(embed_add, x_after_dropout, p_after_dropout, batch_size, seq_len, state);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("6 :tensor+matrix time: %f ms\n", used_time);
#endif
        
        for(int i=0;i<num_layers;i++){
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            layernorm_gpu<<<m, n>>>(norm_a+(i*m*n), norm_a_mean+(i*m), norm_a_rstd+(i*m), embed_add+(i*m*n), d_norm_ag+(i*n), d_norm_ab+(i*n), batch_size, seq_len, state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :layernorm a time in %d layer: %f ms\n", 7+i*22, i+1, used_time);
#endif


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        state, m, state,
                        &alpha, 
                        d_qw+(i*n*n), state,
                        norm_a+(i*m*n), state,
                        &beta, 
                        q_out, state);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                        state, (batch_size*seq_len), state, 
                        &alpha, 
                        d_kw+(i*n*n), state, 
                        norm_a+(i*m*n), state, 
                        &beta, 
                        k_out, state);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                        state, (batch_size*seq_len), state, 
                        &alpha, 
                        d_vw+(i*n*n), state, 
                        norm_a+(i*m*n), state, 
                        &beta, 
                        v_out, state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :q&k&v dense mul time in %d layer: %f ms\n", 8+i*22, i+1, used_time);
#endif


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            tensor_add_vector_gpu<<<m, n>>>(q_bias_out, q_out, d_qb+(i*n), batch_size, seq_len, state);
            tensor_add_vector_gpu<<<m, n>>>(k_bias_out, k_out, d_kb+(i*n), batch_size, seq_len, state);
            tensor_add_vector_gpu<<<m, n>>>(v_bias_out, v_out, d_vb+(i*n), batch_size, seq_len, state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :q&k&v bias time in %d layer: %f ms\n", 9+i*22, i+1, used_time);
#endif
            

            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            transpose_0123to0213_gpu<<<(batch_size*head_num*seq_len), size_per_head>>>(q_bias_out, q_reshape+(i*m*n), batch_size, seq_len, head_num, size_per_head);
            transpose_0123to0213_gpu<<<(batch_size*head_num*seq_len), size_per_head>>>(k_bias_out, k_reshape+(i*m*n), batch_size, seq_len, head_num, size_per_head);
            transpose_0123to0213_gpu<<<(batch_size*head_num*seq_len), size_per_head>>>(v_bias_out, v_reshape+(i*m*n), batch_size, seq_len, head_num, size_per_head);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :q&k&v reshape time in %d layer: %f ms\n", 10+i*22, i+1, used_time);
#endif
            
            offset = 0;
            offset1 = 0;

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            for(int j=0;j<batch_size*head_num;j++){

                hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                seq_len, seq_len, size_per_head, 
                &alpha, 
                k_reshape+(i*m*n)+offset, size_per_head, 
                q_reshape+(i*m*n)+offset, size_per_head, 
                &beta, 
                qk_out+offset1, seq_len);
                offset+= (seq_len*size_per_head);
                offset1 += (seq_len*seq_len);
            }

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :q*kT time in %d layer: %f ms\n", 11+i*22, i+1, used_time);
#endif  


#ifdef DRAW_MAT

        // generate_qk_mask(mask, 10, seq_len);
        // generate_qk_mask_big_bird(mask, seq_len, 32, 64);
        generate_qk_mask_fixed(mask, seq_len, 4);
        hipMemset(d_mask, 0.0, sizeof(int)*seq_len*seq_len);
        hipMemcpy(d_mask, mask, sizeof(int)*seq_len*seq_len, hipMemcpyHostToDevice);

#endif




#ifdef DRAW_MAT

        
        hipMemcpy(qk_out_cpu, qk_out, sizeof(float)*batch_size*head_num*seq_len*seq_len, hipMemcpyDeviceToHost);

        FILE *fp = NULL;

        fp = fopen("fixed_mask_after_qk.txt", "w");

        if(fp == NULL){

                printf("open error!\n");
        }

        for(int l = 0;l<head_num*batch_size;l++){
            for(int j=0;j<seq_len;j++){
                for(int k=0;k<seq_len;k++){
                    qk_out_cpu[l*seq_len*seq_len+j*seq_len+k] *= mask[j*seq_len+k];
                    // if(j<k){
                    //     qk_out_cpu[l*seq_len*seq_len+j*seq_len+k] = 0.0;
                    // }
                    fprintf(fp, "%.25f ", qk_out_cpu[l*seq_len*seq_len+j*seq_len+k]);
                }
            }
        }

        fclose(fp);
        
        return 0;

#endif


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            // softmax_gpu<<<(batch_size*head_num), seq_len>>>(softmax_out+(i*batch_size*head_num*seq_len*seq_len), qk_out, scaler, batch_size, head_num, seq_len);
            softmax_gpu_with_mask<<<(batch_size*head_num), seq_len>>>(softmax_out+(i*batch_size*head_num*seq_len*seq_len), qk_out, d_mask, scaler, batch_size, head_num, seq_len);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :softmax time in %d layer: %f ms\n", 12+i*22, i+1, used_time);
#endif

                
#ifdef DRAW_MAT  

        hipMemcpy(softmax_out_cpu, softmax_out, sizeof(float)*batch_size*head_num*seq_len*seq_len, hipMemcpyDeviceToHost);

        // FILE *fp1 = NULL;

        fp = fopen("bb_mask_softmax.txt", "w");

        if(fp == NULL){

                printf("open error!\n");
        }

        for(int j=0;j<batch_size*head_num*seq_len*seq_len;j++){

            fprintf(fp, "%.15f ", softmax_out_cpu[j]);
            
        }

        fclose(fp);

#endif

        
            offset = 0;
            offset1 = 0;
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            for(int j=0;j<batch_size*head_num;j++){
                
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                        size_per_head, seq_len, seq_len, 
                        &alpha, 
                        v_reshape+(i*m*n)+offset, size_per_head, 
                        softmax_out+offset1,seq_len, 
                        &beta, 
                        sv_out+offset, size_per_head);
 
                offset1+= (seq_len*seq_len);
                offset += (seq_len*size_per_head); 
            }
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :qk*v time in %d layer: %f ms\n", 13+i*22, i+1, used_time);
#endif


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            transpose_0213to0123_gpu<<<(batch_size*head_num*seq_len), size_per_head>>>(sv_out, sv_out_reshape+(i*m*n), batch_size, seq_len, head_num, size_per_head);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :v reshape time in %d layer: %f ms\n", 14+i*22, i+1, used_time);
#endif            
            


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                        state, (batch_size*seq_len), state, 
                        &alpha, 
                        d_aw+(i*n*n), state, 
                        sv_out_reshape+(i*m*n),state, 
                        &beta, 
                        a_out, state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :a dense mul time in %d layer: %f ms\n", 15+i*22, i+1, used_time);
#endif


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            tensor_add_vector_gpu<<<m, n>>>(a_bias_out, a_out, d_ab+(i*n), batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :a bias time in %d layer: %f ms\n", 16+i*22, i+1, used_time);
#endif

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            
        GendropoutMask<<<m, n>>>(d_entropy, d_amask+i*seq_len*n, probe, seq_len*state); 

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :a gen mask time in %d layer: %f ms\n", 17+i*22, i+1, used_time);
#endif

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            ApplydropoutMask<<<320, 512>>>(a_after_dropout, a_bias_out, d_amask+i*seq_len*n, probe, seq_len*state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :a dropout time in %d layer: %f ms\n", 18+i*22, i+1, used_time);
#endif
            


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            tensor_add_tensor_gpu<<<m, n>>>(add_1+(i*m*n), embed_add+(i*m*n), a_after_dropout, batch_size, seq_len, state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :tensor+tensor time in %d layer: %f ms\n", 19+i*22, i+1, used_time);
#endif            
            


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            layernorm_gpu<<<m, n>>>(norm_m+(i*m*n), norm_m_mean+(i*m), norm_m_rstd+(i*m), add_1+(i*m*n), d_norm_mg+(i*n), d_norm_mb+(i*n), batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :layernorm m time in %d layer: %f ms\n", 20+i*22, i+1, used_time);
#endif           
            

            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                        n*4, m, n, 
                        &alpha, 
                        d_m1w+(i*n*n*4), n*4, 
                        norm_m+(i*n*m), n, 
                        &beta, 
                        m1_out, n*4);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m1 dense mul time in %d layer: %f ms\n", 21+i*22, i+1, used_time);
#endif
            

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            tensor_add_vector_gpu_2048<<<m, n>>>(m1_bias_out+(i*m*n*4), m1_out, d_m1b+(i*n*4), batch_size, seq_len, n*4);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m1 bias time in %d layer: %f ms\n", 22+i*22, i+1, used_time); 
#endif

            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            gelu_gpu<<<m, n>>>(m1_gelu_out+(i*m*n*4), m1_bias_out+(i*m*n*4), batch_size, seq_len, state*4);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m1 gelu time in %d layer: %f ms\n", 23+i*22, i+1, used_time);
#endif           
            
 
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                        state, (batch_size*seq_len), state*4, 
                        &alpha, 
                        d_m2w+(i*n*n*4), state, 
                        m1_gelu_out+(i*m*n*4), state*4, 
                        &beta, 
                        m2_out, state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m2 dense mul time in %d layer: %f ms\n", 24+i*22, i+1, used_time);
#endif

            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            tensor_add_vector_gpu<<<m, n>>>(m2_bias_out, m2_out, d_m2b+(i*n), batch_size, seq_len, state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m2 bias time in %d layer: %f ms\n", 25+i*22, i+1, used_time);
#endif

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            
        GendropoutMask<<<m, n>>>(d_entropy, d_mmask+i*seq_len*n, probe, seq_len*state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m2 gen mask time in %d layer: %f ms\n", 26+i*22, i+1, used_time);
#endif
          
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            ApplydropoutMask<<<320, 512>>>(m_after_dropout, m2_bias_out, d_mmask+i*seq_len*n, probe, seq_len*state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m2 dropout time in %d layer: %f ms\n", 27+i*22, i+1, used_time);
#endif            

            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            tensor_add_tensor_gpu<<<m, n>>>(embed_add+(i+1)*m*n, add_1+(i*m*n), m_after_dropout, batch_size, seq_len, state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :tensor+tensor time in %d layer: %f ms\n", 28+i*22, i+1, used_time);
#endif
            

        }

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                    vocab_size, m, n,
                    &alpha,
                    d_xembed, n,
                    embed_add+(num_layers*m*n), n,
                    &beta,
                    logits, vocab_size);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("139 :logits dense mul time: %f ms\n", used_time); 
#endif        

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        softmax_cross_entropy_with_logits_gpu<<<m, 256>>>(softmax_logits, d_loss, logits, d_ylabel, batch_size, seq_len, vocab_size);
        
        float *loss_val;
        hipMalloc((void **)&loss_val, sizeof(float));

        loss_add<<<1, n>>>(loss_val, d_loss, m);

        float *loss_c = (float*)malloc(sizeof(float));
        hipMemcpy(loss_c, loss_val, sizeof(float), hipMemcpyDeviceToHost);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("140 :calculate loss time: %f ms\n", used_time); 
#endif

        // printf("%f \n", loss_c[0]);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        cross_entropy_grad_gpu<<<m, vocab_size>>>(entropy_grad, softmax_logits, batch_size, seq_len, vocab_size);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("141 :cross entropy grad time: %f ms\n", used_time);
#endif


#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    n, m, vocab_size,
                    &alpha,
                    d_xembed, n,
                    entropy_grad, vocab_size,
                    &beta,
                    logits_dx+(num_layers*m*n), n);

        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                    vocab_size, n, m,
                    &alpha,
                    entropy_grad, vocab_size,
                    embed_add+(num_layers*m*n), n,
                    &beta,
                    logits_dw, vocab_size);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("142 :logits grad(dx&dw) time: %f ms\n", used_time); 
#endif


        for(int i = num_layers-1; i>=0; i--){
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            ApplydropoutMask<<<320, 512>>>(dx1, logits_dx+((i+1)*m*n), d_mmask+(i*seq_len*n), probe, seq_len*state); 

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m dropout grad time in %d layer: %f ms\n", 143+((num_layers-1)-i)*25, i+1, used_time);
#endif    


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            bias_grad_db_gpu<<<seq_len, n>>>(d_m2db+(i*n), dx1, batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m2 grad(bias) time in %d layer: %f ms\n", 144+((num_layers-1)-i)*25, i+1, used_time);
#endif
            gradients_add_gpu_512<<<1, 512>>>(gradients_sum+(17+i*16), d_m2db+(i*n), state);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        state*4, m, n, 
                        &alpha,
                        d_m2w+(i*n*n*4), n,
                        dx1, n,
                        &beta,
                        dx_41, state*4);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        state, state*4, m, 
                        &alpha,
                        dx1, n,
                        m1_gelu_out+(i*m*n*4), n*4,
                        &beta,
                        d_m2dw+(i*n*n*4), n); 

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m2 grad(dx&dw) time in %d layer: %f ms\n", 145+((num_layers-1)-i)*25, i+1, used_time);
#endif
            gradients_add_gpu_512_2048<<<2048, 512>>>(gradients_sum+(16+i*16), d_m2dw+(i*n*n*4), n*n*4);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            gelu_grad_gpu<<<m, n>>>(dx_42, dx_41, m1_bias_out+(i*m*n*4), d_m1b+(i*n*4), batch_size, seq_len, state*4);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m1 grad(gelu) time in %d layer: %f ms\n", 146+((num_layers-1)-i)*25, i+1, used_time);
#endif


            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            bias_grad_db_2048_gpu<<<seq_len, n>>>(d_m1db+(i*n*4), dx_42, batch_size, seq_len, state*4);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m1 grad(bias) time in %d layer: %f ms\n", 147+((num_layers-1)-i)*25, i+1, used_time);
#endif

            gradients_add_gpu_2048<<<4, 512>>>(gradients_sum+(15+(i*16)), d_m1db+(i*n*4), n*4);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif            
            hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        n, m, n*4, 
                        &alpha,
                        d_m1w+(i*n*n*4), n*4,
                        dx_42, n*4,
                        &beta,
                        dx1, n);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        n*4, n, m, 
                        &alpha,
                        dx_42, n*4,
                        norm_m+(i*m*n), n,
                        &beta,
                        d_m1dw+(i*n*n*4), n*4);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :m1 grad(dx&dw) time in %d layer: %f ms\n", 148+((num_layers-1)-i)*25, i+1, used_time);
#endif 
            
            gradients_add_gpu_512_2048<<<2048, 512>>>(gradients_sum+(14+i*16), d_m1dw+(i*n*n*4), n*n*4);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            layernorm_dg_db_gpu<<<seq_len, n>>>(d_norm_mdg+(i*n), d_norm_mdb+(i*n), dx1, add_1+(i*m*n), norm_m_mean+(i*m), norm_m_rstd+(i*m), batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :layernorm grad(db&dg) time in %d layer: %f ms\n", 149+((num_layers-1)-i)*25, i+1, used_time);
#endif

            gradients_add_gpu_512<<<1, 512>>>(gradients_sum+(13+i*16), d_norm_mdg+(i*n), n);
            gradients_add_gpu_512<<<1, 512>>>(gradients_sum+(12+i*16), d_norm_mdb+(i*n), n);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            layernorm_grad_dx_gpu<<<m, n>>>(dx2, add_1+(i*m*n), norm_m_mean+(i*m), norm_m_rstd+(i*m), dx1, d_norm_ag+(i*n), batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :layernorm grad(dx) time in %d layer: %f ms\n", 150+((num_layers-1)-i)*25, i+1, used_time);
#endif
            

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            tensor_add_tensor_gpu<<<m, n>>>(grad_add1, logits_dx+((i+1)*m*n), dx2, batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :tensor+tensor time in %d layer: %f ms\n", 151+((num_layers-1)-i)*25, i+1, used_time);
#endif
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            ApplydropoutMask<<<320, 512>>>(dx2, grad_add1, d_amask+(i*seq_len*n), probe, seq_len*state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :a dropout grad time in %d layer: %f ms\n", 152+((num_layers-1)-i)*25, i+1, used_time);  
#endif
            
                  
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            bias_grad_db_gpu<<<320, n>>>(d_adb+(i*n), dx2, batch_size, seq_len, state);   
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :a grad(bias) time in %d layer: %f ms\n", 153+((num_layers-1)-i)*25, i+1, used_time); 
#endif
            gradients_add_gpu_512<<<1, 512>>>(gradients_sum+(11+i*16), d_adb+(i*n), n);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        n, m, n, 
                        &alpha,
                        d_aw+(i*n*n), n,
                        dx2, n,
                        &beta,
                        dx1, n);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        n, n, m, 
                        &alpha,
                        dx2, n,
                        sv_out_reshape+(i*m*n), n,
                        &beta,
                        d_adw+(i*n*n), n);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :a grad(dx&dw) time in %d layer: %f ms\n", 154+((num_layers-1)-i)*25, i+1, used_time); 
#endif

            gradients_add_gpu_512_512<<<512,512>>>(gradients_sum+(10+i*16), d_adw+(i*n*n), n*n);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            transpose_0123to0213_gpu<<<(batch_size*head_num*seq_len), size_per_head>>>(dx1, dx2, batch_size, seq_len, head_num, size_per_head);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :a reshape time in %d layer: %f ms\n", 155+((num_layers-1)-i)*25, i+1, used_time); 
#endif
            
            offset = 0;
            offset1 = 0;

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            for(int j=0; j<batch_size*head_num;j++){
                
                hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                            seq_len, seq_len, size_per_head,
                            &alpha,
                            v_reshape+(i*m*n)+offset, size_per_head, 
                            dx2+offset, size_per_head,
                            &beta,
                            sv_grad+offset1, seq_len);

                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                            size_per_head, seq_len, seq_len,
                            &alpha,
                            dx2+offset, size_per_head, 
                            softmax_out+(i*batch_size*head_num*seq_len*seq_len)+offset1, seq_len,
                            &beta,
                            v_grad+offset, size_per_head);

                offset += seq_len*size_per_head;
                offset1 += seq_len*seq_len;

            }

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :sv grad(dx&dw) time in %d layer: %f ms\n", 156+((num_layers-1)-i)*25, i+1, used_time);
#endif
 

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            softmax_grad_gpu<<<(batch_size*head_num), seq_len>>>(softmaxgrad, sv_grad, softmax_out+(i*batch_size*head_num*seq_len*seq_len), scaler, batch_size, head_num, seq_len);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :softmax grad time in %d layer: %f ms\n", 157+((num_layers-1)-i)*25, i+1, used_time);
#endif
            

            offset = 0;
            offset1 = 0;
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            for(int j=0;j<batch_size*head_num;j++){

                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                            size_per_head, seq_len, seq_len,
                            &alpha,
                            k_reshape+(i*m*n)+offset, size_per_head, 
                            softmaxgrad+offset1, seq_len,
                            &beta,
                            k_grad+offset, size_per_head);

                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                            seq_len, size_per_head, seq_len,
                            &alpha,
                            softmaxgrad+offset1, seq_len, 
                            q_reshape+(i*m*n)+offset, size_per_head,
                            &beta,
                            q_grad+offset, seq_len);
                
                offset += seq_len*size_per_head;
                offset1 += seq_len*seq_len;
            }

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :qk grad(dx&dw) time in %d layer: %f ms\n", 158+((num_layers-1)-i)*25, i+1, used_time);
#endif


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            transpose_0123to0132_gpu<<<(batch_size*head_num*seq_len), size_per_head>>>(q_grad, q_grad_trans, batch_size, seq_len, head_num, size_per_head);
            transpose_0213to0123_gpu<<<(batch_size*seq_len*head_num), size_per_head>>>(q_grad_trans, q_grad_reshape, batch_size, seq_len, head_num, size_per_head);
            transpose_0213to0123_gpu<<<(batch_size*seq_len*head_num), size_per_head>>>(k_grad, k_grad_reshape, batch_size, seq_len, head_num, size_per_head);
            transpose_0213to0123_gpu<<<(batch_size*seq_len*head_num), size_per_head>>>(v_grad, v_grad_reshape, batch_size, seq_len, head_num, size_per_head);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :q&k&v grad reshape time in %d layer: %f ms\n", 159+((num_layers-1)-i)*25, i+1, used_time);
#endif
            


#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            bias_grad_db_gpu<<<320, n>>>(d_qdb+(i*n), q_grad_reshape, batch_size, seq_len, state);
            bias_grad_db_gpu<<<320, n>>>(d_kdb+(i*n), k_grad_reshape, batch_size, seq_len, state);
            bias_grad_db_gpu<<<320, n>>>(d_vdb+(i*n), v_grad_reshape, batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :q&k&v grad(bias) time in %d layer: %f ms\n", 160+((num_layers-1)-i)*25, i+1, used_time);
#endif
            
            gradients_add_gpu_512<<<1,512>>>(gradients_sum+(9+i*16), d_qdb+(i*n), n);
            gradients_add_gpu_512<<<1,512>>>(gradients_sum+(8+i*16), d_kdb+(i*n), n);
            gradients_add_gpu_512<<<1,512>>>(gradients_sum+(7+i*16), d_vdb+(i*n), n);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        n, m, n, 
                        &alpha,
                        d_vw+(i*n*n), n,
                        v_grad_reshape, n,
                        &beta,
                        dx1, n);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        n, n, m, 
                        &alpha,
                        v_grad_reshape, n,
                        norm_a+(i*m*n), n,
                        &beta,
                        d_vdw+(i*n*n), n);  

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :v grad(dx&dw) time in %d layer: %f ms\n", 161+((num_layers-1)-i)*25, i+1, used_time);
#endif
            
            gradients_add_gpu_512_512<<<512,512>>>(gradients_sum+(6+i*16), d_vdw+(i*n*n), n*n);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        n, m, n, 
                        &alpha,
                        d_kw+(i*n*n), n,
                        k_grad_reshape, n,
                        &beta,
                        dx2, n);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        n, n, m, 
                        &alpha,
                        k_grad_reshape, n,
                        norm_a+(i*m*n), n,
                        &beta,
                        d_kdw+(i*n*n), n); 

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :k grad(dx&dw) time in %d layer: %f ms\n", 162+((num_layers-1)-i)*25, i+1, used_time);
#endif 
            gradients_add_gpu_512_512<<<512,512>>>(gradients_sum+(5+i*16), d_kdw+(i*n*n), n*n);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        n, m, n, 
                        &alpha,
                        d_qw+(i*n*n), n,
                        q_grad_reshape, n,
                        &beta,
                        dx3, n);

            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        n, n, m, 
                        &alpha,
                        q_grad_reshape, n,
                        norm_a+(i*m*n), n,
                        &beta,
                        d_qdw+(i*n*n), n); 

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :q grad(dx&dw) time in %d layer: %f ms\n", 163+((num_layers-1)-i)*25, i+1, used_time);
#endif

            gradients_add_gpu_512_512<<<512,512>>>(gradients_sum+(4+i*16), d_qdw+(i*n*n), n*n);

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
        //     tensor_add_tensor_gpu<<<m, n>>>(v_grad, dx1, dx2, batch_size, seq_len, state);
        //     tensor_add_tensor_gpu<<<m, n>>>(dx1, v_grad, dx3, batch_size, seq_len, state);
        tensor_add_tensor_add_tensor_gpu<<<m, n>>>(v_grad, dx1, dx2, dx3, batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :(tensor+tensor)x2 time in %d layer: %f ms\n", 164+((num_layers-1)-i)*25, i+1, used_time);
#endif
            

#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif
            layernorm_dg_db_gpu<<<seq_len, n>>>(d_norm_adg+(i*n), d_norm_adb+(i*n), v_grad, embed_add+(i*m*n), norm_a_mean+(i*m), norm_a_rstd+(i*n), batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :layernorm grad(dg&db) time in %d layer: %f ms\n", 165+((num_layers-1)-i)*25, i+1, used_time);
#endif
            
            gradients_add_gpu_512<<<1, 512>>>(gradients_sum+(3+i*16), d_norm_adg+(i*n), n);
            gradients_add_gpu_512<<<1, 512>>>(gradients_sum+(2+i*16), d_norm_adb+(i*n), n);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif     
            layernorm_grad_dx_gpu<<<m, n>>>(dx2, embed_add+(i*m*n), norm_a_mean+(i*m), norm_a_rstd+(i*m), v_grad, d_norm_ag+(i*n), batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :layernorm grad(dx) time in %d layer: %f ms\n", 166+((num_layers-1)-i)*25, i+1, used_time);
#endif   
            

            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            time_check_begin();
#endif 
            tensor_add_tensor_gpu<<<m, n>>>(logits_dx+(i*m*n), grad_add1, dx2, batch_size, seq_len, state);
            
#ifdef TRANSFORMER_DEBUG
            hipDeviceSynchronize();
            used_time = time_check_end();
            printf("%d :tensor+tensor time in %d layer: %f ms\n", 167+((num_layers-1)-i)*25, i+1, used_time);
#endif 
            
        }
        
#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        ApplydropoutMask<<<320, 512>>>(dx1, logits_dx, d_xmask, probe, seq_len*state);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("293 :x dropout grad time: %f ms\n", used_time);
#endif


#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        add_grad_gpu<<<seq_len,n>>>(p_embed_grad, dx1, batch_size, seq_len, state);

        ApplydropoutMask_SmallSize<<<10,512>>>(p_embed_dw, p_embed_grad, d_pmask, probe, 5120);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("294 :p dropout grad time: %f ms\n", used_time);
#endif
        gradients_add_gpu_512_512<<<320, 512>>>(gradients_sum+1, p_embed_dw, seq_len*state);
        
#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        embedding_lookup_grad_gpu<<<m, n>>>(embed_grad, dx1, d_xlabel, batch_size, seq_len, state, vocab_size);
        
#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("295 :embedding grad time: %f ms\n", used_time);
#endif
        


#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        transpose_0123to0132_gpu<<<vocab_size, n>>>(logits_dw, logits_dw_trans, 1, vocab_size, 1, state);
        
#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("296 :logits dw trans time: %f ms\n", used_time);
#endif
        

        
#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        tensor_add_tensor_gpu<<<vocab_size, n>>>(embed_add_out, logits_dw_trans, embed_grad, 1, vocab_size, state);
        
#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("297 :tensor+tensor time: %f ms\n", used_time);
#endif
        
        gradients_add_gpu_512_512<<<256, 512>>>(gradients_sum, embed_add_out, vocab_size*state);

        for(int i=0;i<(num_layers*16+2);i++){
            hipMemcpy(gradients_sumc+i, gradients_sum+i, sizeof(float), hipMemcpyDeviceToHost);
            printf("%.25f %d\n", gradients_sumc[i], i);
            grad_sum_sum += gradients_sumc[i];
        }


        global_norm = sqrt(grad_sum_sum);
        norm_scale = clip_by_global_norm(global_norm, clip_norm);

        // printf("global_norm:%.10f\n", global_norm);
        // printf("norm_sacle:%.10f\n", norm_scale);

        lr = global_step * (1.0/1000) < 1 ? global_step * (1.0/1000) : 1;
        lr *= learning_rate;

        beta1_power = adam_got_beta_power(beta1, global_step);
        beta2_power = adam_got_beta_power(beta2, global_step);

        float lr_2 = adam_got_lr(lr, beta1_power, beta2_power);

#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        time_check_begin();
#endif
        adam_apply_gradients_gpu<<<1, n>>>(d_norm_ag, d_norm_adg, d_norm_a_g_mt, d_norm_a_g_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2, 
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*num_layers);

        adam_apply_gradients_gpu<<<1, n>>>(d_norm_ab, d_norm_adb, d_norm_a_b_mt, d_norm_a_b_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*num_layers);

        adam_apply_gradients_gpu<<<1, n>>>(d_norm_mg, d_norm_mdg, d_norm_m_g_mt, d_norm_m_g_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*num_layers);

        adam_apply_gradients_gpu<<<1, n>>>(d_qb, d_qdb, d_q_b_mt, d_q_b_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*num_layers);

        adam_apply_gradients_gpu<<<1, n>>>(d_kb, d_kdb, d_k_b_mt, d_k_b_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*num_layers);

        adam_apply_gradients_gpu<<<1, n>>>(d_vb, d_vdb, d_v_b_mt, d_v_b_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*num_layers);

        adam_apply_gradients_gpu<<<1, n>>>(d_ab, d_adb, d_a_b_mt, d_a_b_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*num_layers);

        adam_apply_gradients_gpu<<<4, n>>>(d_m1b, d_m1db, d_m1_b_mt, d_m1_b_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*4*num_layers);

        adam_apply_gradients_gpu<<<1, n>>>(d_m2b, d_m2db, d_m2_b_mt, d_m2_b_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*num_layers);

        adam_apply_gradients_gpu<<<n, n>>>(d_qw, d_qdw, d_q_w_mt, d_q_w_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*state*num_layers);

        adam_apply_gradients_gpu<<<n, n>>>(d_kw, d_kdw, d_k_w_mt, d_k_w_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*state*num_layers);

        adam_apply_gradients_gpu<<<n, n>>>(d_vw, d_vdw, d_v_w_mt, d_v_w_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*state*num_layers); 

        adam_apply_gradients_gpu<<<n, n>>>(d_aw, d_adw, d_a_w_mt, d_a_w_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*state*num_layers);   

        adam_apply_gradients_gpu<<<n*4, n>>>(d_m1w, d_m1dw, d_m1_w_mt, d_m1_w_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*state*4*num_layers);

        adam_apply_gradients_gpu<<<n*4, n>>>(d_m2w, d_m2dw, d_m2_w_mt, d_m2_w_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, state*state*4*num_layers); 

        adam_apply_gradients_gpu<<<seq_len, n>>>(d_pembed, p_embed_dw, d_p_embed_mt, d_p_embed_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, seq_len*state*num_layers); 

        adam_apply_gradients_gpu<<<vocab_size, n>>>(d_xembed, embed_add_out, d_x_embed_mt, d_x_embed_vt, 
                                        beta1, beta2, beta1_power, beta2_power, lr_2,
                                        epsilon, norm_scale, grad_scale, clip_sigma, vocab_size*state*num_layers);
        
#ifdef TRANSFORMER_DEBUG
        hipDeviceSynchronize();
        used_time = time_check_end();
        printf("298 :Adam update time: %f ms\n", used_time);
#endif
        
    }

 


    return 0;
}